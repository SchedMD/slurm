// Slurm regression test40.8.prog.cu

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <sys/time.h>
// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
	int index = threadIdx.x;
	int stride = blockDim.x;
	for (int i = index; i < n; i += stride)
		y[i] = x[i] + y[i];
}

int main(void)
{
	int N = 1024 * 1024 * 16;
	int i;
	float *x, *y;
	float maxError = 0.0f;
	struct timeval tv1, tv2;
	int delta_t;

	// Get start time
	gettimeofday(&tv1, NULL);

	// Allocate Unified Memory – accessible from CPU or GPU
	if (hipMallocManaged(&x, N * sizeof(float)) != hipSuccess) {
	        std::cerr << "Couldn't allocate memory for x: " << errno << std::endl;
		return 1;
	}
	if (hipMallocManaged(&y, N * sizeof(float)) != hipSuccess) {
	        std::cerr << "Couldn't allocate memory for y: " << errno << std::endl;
		return 1;
	}

	// initialize x and y arrays on the host
	for (i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	// Run kernel on 256 elements at a time on the GPU
	add<<<1, 256>>>(N, x, y);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// Check for errors (all values should be 3.0f)
	for (i = 0; i < N; i++)
		maxError = fmax(maxError, fabs(y[i] - 3.0f));
	std::cout << "Max error: " << maxError << std::endl;

	// Free memory
	hipFree(x);
	hipFree(y);

	// Get start time
	gettimeofday(&tv2, NULL);
	delta_t  = (tv2.tv_sec  - tv1.tv_sec) * 1000000;
	delta_t += (tv2.tv_usec - tv1.tv_usec);
	std::cout << "Run Time (usec): " << delta_t << std::endl;

	return 0;
}
